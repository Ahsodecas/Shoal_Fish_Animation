#include "hip/hip_runtime.h"
﻿#pragma once
#include "definitions.h"


const char* vertexShaderSource = R"(
#version 330 core
in vec2 position;

void main()
{
    gl_Position = vec4(position, 0.0, 1.0);
})";

const char* fragmentShaderSource = R"(
#version 330 core
out vec4 FragColor;
void main() {
    FragColor = vec4(0.1, 0.6, 0.9, 1.0); // Light blue
}
)";


void calculateTriangleVerticesCPU(float* positions, BoidsVelocity boidsVelocity, int num_boids);
int oneIterationCPU(float** boids_positions, BoidsVelocity* boidsVelocity);
void updateBoidsPositionCPU(float* positions, BoidsVelocity boidsVelocity, int numBoids, float dt);
void updateBoidsVelocityCPU(float* positions, BoidsVelocity boidsVelocity, int numBoids, float dt, bool cursorOverWindow, double cursorX, double cursorY);


__global__ void updateBoidsVelocity(float* positions, BoidsVelocity boidsVelocity, int numBoids, float dt, bool cursorOverWindow, double cursorX, double cursorY)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numBoids) return;
    float close_dx = 0, close_dy = 0;
    float xvel_avg = 0, yvel_avg = 0, xpos_avg = 0, ypos_avg = 0;
    int neighbors = 0;

    float my_x = positions[6 * idx];
    float my_y = positions[6 * idx + 1];
    float my_vx = boidsVelocity.vx[idx];
    float my_vy = boidsVelocity.vy[idx];
    my_x = ((positions[6 * idx] + 1.0f) / 2.0f) * SCREEN_WIDTH;
    my_y = (1.0f - ((positions[6 * idx + 1] + 1.0f) / 2.0f)) * SCREEN_HEIGHT;

    for (int i = 0; i < numBoids; i++) 
    {
        if (i == idx) continue;
        float x = ((positions[6 * i] + 1.0f) / 2.0f) * SCREEN_WIDTH;
        float y = (1.0f - ((positions[6 * i + 1] + 1.0f) / 2.0f)) * SCREEN_HEIGHT;
        float dx = x - my_x;
        float dy = y - my_y;
        float dist = sqrt(dx * dx + dy * dy);

        // Separation
        if (dist < PROTECTED_RANGE) 
        { 
            close_dx -= dx;
            close_dy -= dy;
        }
        // Alignment and Cohesion
        if (dist < VISUAL_RANGE) 
        { 
            xvel_avg += boidsVelocity.vx[i];
            yvel_avg += boidsVelocity.vy[i];
            xpos_avg += x;
            ypos_avg += y;
            neighbors++;
        }
    }

    // Calculate alignment and cohesion
    if (neighbors > 0) 
    {
        xvel_avg /= neighbors;
        yvel_avg /= neighbors;
        xpos_avg /= neighbors;
        ypos_avg /= neighbors;

        // Alignment
        my_vx += (xvel_avg - my_vx) * MATCHING_FACTOR;   
        my_vy += (yvel_avg - my_vy) * MATCHING_FACTOR;

        // Cohesion
        my_vx += (xpos_avg - my_x) * CENTERING_FACTOR;   
        my_vy += (ypos_avg - my_y) * CENTERING_FACTOR;
    }

    // Separation
    my_vx += close_dx * AVOID_FACTOR;  
    my_vy += close_dy * AVOID_FACTOR;

    // Avoid cursor
    if (cursorOverWindow)
    {
        float dx_cursor = cursorX - my_x;
        float dy_cursor = cursorY - my_y;
        float dist_cursor = sqrtf(dx_cursor * dx_cursor + dy_cursor * dy_cursor);
        if (dist_cursor < VISUAL_RANGE)
        {
            my_vx -= dx_cursor * CURSOR_AVOID_FACTOR;
            my_vy -= dy_cursor * CURSOR_AVOID_FACTOR;
        }
    }

    // Bias 
    // biased to the right
    if (idx % 419 == 0)
    {
        my_vx = (1 - BIAS) * my_vx + (BIAS * 1);
    } 
    // biased to the left
    else if (idx % 409 == 0)
    {
        my_vx = (1 - BIAS) * my_vx + (BIAS * (-1));
    }
        
    // Edge Avoidance
    if (my_x < EDGE_MARGIN) my_vx += TURN_FACTOR;
    if (my_x > SCREEN_WIDTH - EDGE_MARGIN) my_vx -= TURN_FACTOR;
    if (my_y < EDGE_MARGIN) my_vy += TURN_FACTOR;
    if (my_y > SCREEN_HEIGHT - EDGE_MARGIN) my_vy -= TURN_FACTOR;

    // Speed Limits
    float speed = sqrt(my_vx * my_vx + my_vy * my_vy);
    if (speed < MIN_SPEED) 
    {
        my_vx = (my_vx / speed) * MIN_SPEED;
        my_vy = (my_vy / speed) * MIN_SPEED;
    }
    if (speed > MAX_SPEED)
    {
        my_vx = (my_vx / speed) * MAX_SPEED;
        my_vy = (my_vy / speed) * MAX_SPEED;
    }

    my_x += my_vx * dt;
    my_y += my_vy * dt;
    boidsVelocity.vx[idx] = my_vx;
    boidsVelocity.vy[idx] = my_vy;
}

__global__ void updateBoidsPosition(float* positions, BoidsVelocity boidsVelocity, int numBoids, float dt)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numBoids) return;
    float my_x = ((positions[6 * idx] + 1.0f) / 2.0f) * SCREEN_WIDTH;
    float my_y = (1.0f - ((positions[6 * idx + 1] + 1.0f) / 2.0f)) * SCREEN_HEIGHT;
    my_x += boidsVelocity.vx[idx] * dt;
    my_y += boidsVelocity.vy[idx] * dt;
    positions[6 * idx] = (my_x * 2) / SCREEN_WIDTH - 1.0f;
    positions[6 * idx + 1] = 1.0f - (my_y * 2) / SCREEN_HEIGHT;
}

void checkShaderCompilation(GLuint shader, std::string type)
{
    GLint success;
    char infoLog[512];
    if (type == "PROGRAM") 
    {
        glGetProgramiv(shader, GL_LINK_STATUS, &success);
        if (!success) 
        {
            glGetProgramInfoLog(shader, 512, NULL, infoLog);
            std::cerr << "ERROR::PROGRAM_LINKING_ERROR: " << infoLog << std::endl;
        }
    }
    else {
        glGetShaderiv(shader, GL_COMPILE_STATUS, &success);
        if (!success) 
        {
            glGetShaderInfoLog(shader, 512, NULL, infoLog);
            std::cerr << "ERROR::SHADER_COMPILATION_ERROR (" << type << "): " << infoLog << std::endl;
        }
    }
}

void processInput(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
    {
        glfwSetWindowShouldClose(window, true);
    }
    if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS)
    {
        Moving = !Moving;
    }
}

void cursorEnterCallback(GLFWwindow* window, int entered)
{
    if (entered) 
    {
        CursorOverWindow = true;
    }
    else {
        CursorOverWindow = false;
    }
}

void cursorPositionCallback(GLFWwindow* window, double xpos, double ypos)
{
    cursorX = xpos;
    cursorY = ypos;
}

void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

hipError_t initBoids(GLuint* VBO, GLuint* VAO, hipGraphicsResource** cudaVBO, BoidsVelocity* boidsVelocity)
{
    srand(time(NULL));
    hipError_t cudaStatus;

    float* temp_positions = (float*)malloc(3 * NUM_BOIDS * 2 * sizeof(float));
    if (temp_positions == NULL)
    {
        fprintf(stderr, "malloc failed!");
        return hipErrorOutOfMemory;
    }
    float* temp_vx = (float*)malloc(NUM_BOIDS * sizeof(float));
    if (temp_vx == NULL)
    {
        fprintf(stderr, "malloc failed!");
        return hipErrorOutOfMemory;
    }
    float* temp_vy = (float*)malloc(NUM_BOIDS * sizeof(float));
    if (temp_vx == NULL)
    {
        fprintf(stderr, "malloc failed!");
        return hipErrorOutOfMemory;
    }

    glGenVertexArrays(1, VAO);
    glGenBuffers(1, VBO);
    glBindVertexArray(*VAO);

    for (int i = 0; i < 3 * NUM_BOIDS * 2; i += 2) 
    {
        temp_positions[i] = ((rand() % SCREEN_WIDTH) / (SCREEN_WIDTH / 2.0f)) - 1.0f; // Normalize X to [-1, 1]
        temp_positions[i + 1] = ((rand() % SCREEN_HEIGHT) / (SCREEN_HEIGHT / 2.0f)) - 1.0f; // Normalize Y to [-1, 1]
    }

    glBindBuffer(GL_ARRAY_BUFFER, *VBO);
    glBufferData(GL_ARRAY_BUFFER, 3 * NUM_BOIDS * 2 * sizeof(float), temp_positions, GL_DYNAMIC_DRAW);

    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 0, 0);
    glEnableVertexAttribArray(0);

    cudaStatus = hipGraphicsGLRegisterBuffer(cudaVBO, *VBO, hipGraphicsRegisterFlagsNone);
    if (cudaStatus != hipSuccess) 
    {
        std::cerr << "Error registering buffer with CUDA: " << hipGetErrorString(cudaStatus) << std::endl;
        return cudaStatus;
    }

    cudaStatus = hipMalloc(&boidsVelocity->vx, NUM_BOIDS * sizeof(float));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }
    cudaStatus = hipMalloc(&boidsVelocity->vy, NUM_BOIDS * sizeof(float));
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    for (int i = 0; i < NUM_BOIDS; i++) 
    {
        temp_vx[i] = ((rand() % 20) - 10) / 10.0f;
        temp_vy[i] = ((rand() % 20) - 10) / 10.0f;
    }

    cudaStatus = hipMemcpy(boidsVelocity->vx, temp_vx, NUM_BOIDS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        return cudaStatus;
    }
    cudaStatus = hipMemcpy(boidsVelocity->vy, temp_vy, NUM_BOIDS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMemcpy failed!");
        return cudaStatus;
    }

    free(temp_vx);
    free(temp_vy);
    free(temp_positions);

    return hipSuccess;
}

void initShader(GLuint* shaderProgram)
{
    GLuint vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);
    checkShaderCompilation(vertexShader, "VERTEX");

    GLuint fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);
    checkShaderCompilation(fragmentShader, "FRAGMENT");

    *shaderProgram = glCreateProgram();
    glAttachShader(*shaderProgram, vertexShader);
    glAttachShader(*shaderProgram, fragmentShader);
    glLinkProgram(*shaderProgram);
    checkShaderCompilation(*shaderProgram, "PROGRAM");

    glDeleteShader(vertexShader);
    glDeleteShader(fragmentShader);
}

__global__ void calculateTriangleVertices(float* positions, BoidsVelocity boidsVelocity, int num_boids)
{
    int boid_index = blockIdx.x * blockDim.x + threadIdx.x;
    int index = boid_index * 6;
    if (boid_index >= num_boids) return;

    float my_x = ((positions[index] + 1.0f) / 2.0f) * SCREEN_WIDTH;
    float my_y = (1.0f - ((positions[index + 1] + 1.0f) / 2.0f)) * SCREEN_HEIGHT;

    // Boids are drawn as isosceles triangles of height 5*sqrt(3) heading in direction pointed by point (positions[index], positions[index + 1])
    float s =  2 * sqrtf(3);
    float triangle_h = 5 * sqrtf(3);



    float vector_length = sqrt(boidsVelocity.vx[boid_index] * boidsVelocity.vx[boid_index] + boidsVelocity.vy[boid_index] * boidsVelocity.vy[boid_index]);
    float h_x = my_x - (triangle_h * (boidsVelocity.vx[boid_index] / vector_length));
    float h_y = my_y - (triangle_h * (boidsVelocity.vy[boid_index] / vector_length));

    float x1 = h_x + ((my_y - h_y) / s);
    float x2 = h_x + ((h_y - my_y) / s);
    float y1 = h_y + ((h_x - my_x) / s);
    float y2 = h_y + ((my_x - h_x) / s);


    positions[index] = positions[index];
    positions[index + 2] = (x1 * 2) / SCREEN_WIDTH - 1.0f;
    positions[index + 4] = (x2 * 2) / SCREEN_WIDTH - 1.0f;

    positions[index + 1] = positions[index + 1];
    positions[index + 3] = 1.0f - (y1 * 2) / SCREEN_HEIGHT;
    positions[index + 5] = 1.0f - (y2 * 2) / SCREEN_HEIGHT;
}



hipError_t oneIteration(hipGraphicsResource** cudaVBO, float ** boids_positions, BoidsVelocity* boidsVelocity, int BLOCKS_NUM)
{
    hipError_t cudaStatus;

    updateBoidsVelocity << <BLOCKS_NUM, BLOCK_SIZE >> > (*boids_positions, *boidsVelocity, NUM_BOIDS, DT, CursorOverWindow, cursorX, cursorY);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "CUDA kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    updateBoidsPosition << <BLOCKS_NUM, BLOCK_SIZE >> > (*boids_positions, *boidsVelocity, NUM_BOIDS, DT);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "CUDA kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    calculateTriangleVertices << <BLOCKS_NUM, BLOCK_SIZE >> > (*boids_positions, *boidsVelocity, NUM_BOIDS);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "CUDA kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }


    return hipSuccess;
}

void cleanUp(GLuint* VBO, GLuint* VAO, BoidsVelocity* boidsVelocity, GLuint* shaderProgram)
{
    glDeleteBuffers(1, VBO);
    glDeleteVertexArrays(1, VAO);
    hipFree(boidsVelocity->vx);
    hipFree(boidsVelocity->vy);
    glDeleteProgram(*shaderProgram);
    glfwTerminate();
}

int main()
{
    hipError_t cudaStatus;
    int BLOCKS_NUM = (NUM_BOIDS + BLOCK_SIZE - 1) / BLOCK_SIZE;

    if (!glfwInit())
    {
        fprintf(stderr, "Failed to initialize the GLFW library\n");
        return -1;
    }
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    GLFWwindow* window = glfwCreateWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "Shoal of Fish", NULL, NULL);
    if (window == NULL)
    {
        fprintf(stderr, "Failed to create GLFW window\n");
        glfwTerminate();
        return -1;
    }
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    glfwMakeContextCurrent(window);
    glfwSetKeyCallback(window, processInput);
    glfwSetCursorEnterCallback(window, cursorEnterCallback);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        fprintf(stderr, "Failed to initialize GLAD\n");
        glfwTerminate();
        return -1;
    }

    glViewport(0, 0, SCREEN_WIDTH, SCREEN_HEIGHT);

    GLuint shaderProgram;
    initShader(&shaderProgram);

    GLuint VBO, VAO;
    hipGraphicsResource* cudaVBO;
    BoidsVelocity boidsVelocity;
    float* boids_positions = NULL;


    cudaStatus = initBoids(&VBO, &VAO, &cudaVBO, &boidsVelocity);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Boids initialization failed: %s\n", hipGetErrorString(cudaStatus));
        cleanUp(&VBO, &VAO, &boidsVelocity, &shaderProgram);
        return -1;
    }
    
    glPointSize(2.0f);

    int iterations = 0;
    clock_t start = clock();
    clock_t end;


    float* vx = NULL;
    float* vy = NULL;
    float* positions = NULL;
    BoidsVelocity boidsVelocityCPU;
    if (!gpuVersion)
    {
        vx = (float*)malloc(sizeof(float) * NUM_BOIDS);
        vy = (float*)malloc(sizeof(float) * NUM_BOIDS);
        hipMemcpy(vx, boidsVelocity.vx, sizeof(float) * NUM_BOIDS, hipMemcpyDeviceToHost);
        hipMemcpy(vy, boidsVelocity.vy, sizeof(float) * NUM_BOIDS, hipMemcpyDeviceToHost);
        boidsVelocityCPU.vx = vx;
        boidsVelocityCPU.vy = vy;
        positions = (float*)malloc(sizeof(float) * NUM_BOIDS * 3 * 2);
    }

    while (!glfwWindowShouldClose(window))
    {

        if (Moving)
        {
            if (CursorOverWindow) 
            {
                glfwGetCursorPos(window, &cursorX, &cursorY);
            }
            cudaStatus = hipGraphicsMapResources(1, &cudaVBO, 0);
            if (cudaStatus != hipSuccess)
            {
                fprintf(stderr, "hipGraphicsMapResources launch failed: %s\n", hipGetErrorString(cudaStatus));
                return cudaStatus;
            }

            cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&boids_positions, NULL, cudaVBO);
            if (cudaStatus != hipSuccess)
            {
                fprintf(stderr, "hipGraphicsResourceGetMappedPointer launch failed: %s\n", hipGetErrorString(cudaStatus));
                return cudaStatus;
            }
            if (gpuVersion)
            {
                cudaStatus = oneIteration(&cudaVBO, &boids_positions, &boidsVelocity, BLOCKS_NUM);
                if (cudaStatus != hipSuccess)
                {
                    fprintf(stderr, "iteration launch failed: %s\n", hipGetErrorString(cudaStatus));
                    cudaStatus = hipGraphicsUnregisterResource(cudaVBO);
                    if (cudaStatus != hipSuccess)
                    {
                        fprintf(stderr, "hipGraphicsUnregisterResource launch failed: %s\n", hipGetErrorString(cudaStatus));
                    }
                    cleanUp(&VBO, &VAO, &boidsVelocity, &shaderProgram);
                    return -1;
                }
            }
            else
            {
                cudaStatus = hipMemcpy(positions, boids_positions, sizeof(float) * NUM_BOIDS * 3 * 2,  hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess)
                {
                    fprintf(stderr, "hipMemcpy launch failed: %s\n", hipGetErrorString(cudaStatus));
                }
                oneIterationCPU(&positions, &boidsVelocityCPU);
                cudaStatus = hipMemcpy(boids_positions, positions, sizeof(float) * NUM_BOIDS * 3 * 2, hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess)
                {
                    fprintf(stderr, "hipMemcpy launch failed: %s\n", hipGetErrorString(cudaStatus));
                }
            }

            cudaStatus = hipGraphicsUnmapResources(1, &cudaVBO, 0);
            if (cudaStatus != hipSuccess)
            {
                fprintf(stderr, "hipGraphicsResourceGetMappedPointer launch failed: %s\n", hipGetErrorString(cudaStatus));
                return cudaStatus;
            }
        }
        glClear(GL_COLOR_BUFFER_BIT);

        glUseProgram(shaderProgram);
        glBindVertexArray(VAO);
        glDrawArrays(GL_TRIANGLES, 0, 3 * NUM_BOIDS);

        glfwSwapBuffers(window);
        glfwPollEvents();

        iterations++;
        if (iterations == ITERATIONS)
        {
            end = clock();
        }

    }

    if (iterations < ITERATIONS)
    {
        end = clock();
    }
    else
    {
        iterations = ITERATIONS;
    }
    float loopTime = (float)(end - start) / CLOCKS_PER_SEC;
    printf("Time taken for %i iterations: %f\n", iterations, loopTime);

    cudaStatus = hipGraphicsUnregisterResource(cudaVBO);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipGraphicsUnregisterResource launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    cleanUp(&VBO, &VAO, &boidsVelocity, &shaderProgram);

    if (!gpuVersion)
    {
        free(boidsVelocityCPU.vx);
        free(boidsVelocityCPU.vy);
        free(positions);
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return -1;
    }

    return 0;
}


void updateBoidsVelocityCPU(float* positions, BoidsVelocity boidsVelocity, int numBoids, float dt, bool cursorOverWindow, double cursorX, double cursorY)
{
    for (int idx = 0; idx < numBoids; idx++)
    {
        float close_dx = 0, close_dy = 0;
        float xvel_avg = 0, yvel_avg = 0, xpos_avg = 0, ypos_avg = 0;
        int neighbors = 0;

        float my_x = positions[6 * idx];
        float my_y = positions[6 * idx + 1];
        float my_vx = boidsVelocity.vx[idx];
        float my_vy = boidsVelocity.vy[idx];
        my_x = ((positions[6 * idx] + 1.0f) / 2.0f) * SCREEN_WIDTH;
        my_y = (1.0f - ((positions[6 * idx + 1] + 1.0f) / 2.0f)) * SCREEN_HEIGHT;

        for (int i = 0; i < numBoids; i++)
        {
            if (i == idx) continue;
            float x = ((positions[6 * i] + 1.0f) / 2.0f) * SCREEN_WIDTH;
            float y = (1.0f - ((positions[6 * i + 1] + 1.0f) / 2.0f)) * SCREEN_HEIGHT;
            float dx = x - my_x;
            float dy = y - my_y;
            float dist = sqrt(dx * dx + dy * dy);

            // Separation
            if (dist < PROTECTED_RANGE)
            {
                close_dx -= dx;
                close_dy -= dy;
            }
            // Alignment and Cohesion
            if (dist < VISUAL_RANGE)
            {
                xvel_avg += boidsVelocity.vx[i];
                yvel_avg += boidsVelocity.vy[i];
                xpos_avg += x;
                ypos_avg += y;
                neighbors++;
            }
        }

        // Calculate alignment and cohesion
        if (neighbors > 0)
        {
            xvel_avg /= neighbors;
            yvel_avg /= neighbors;
            xpos_avg /= neighbors;
            ypos_avg /= neighbors;

            // Alignment
            my_vx += (xvel_avg - my_vx) * MATCHING_FACTOR;
            my_vy += (yvel_avg - my_vy) * MATCHING_FACTOR;

            // Cohesion
            my_vx += (xpos_avg - my_x) * CENTERING_FACTOR;
            my_vy += (ypos_avg - my_y) * CENTERING_FACTOR;
        }

        // Separation
        my_vx += close_dx * AVOID_FACTOR;
        my_vy += close_dy * AVOID_FACTOR;

        // Avoid cursor
        if (cursorOverWindow)
        {
            float dx_cursor = cursorX - my_x;
            float dy_cursor = cursorY - my_y;
            float dist_cursor = sqrtf(dx_cursor * dx_cursor + dy_cursor * dy_cursor);
            if (dist_cursor < VISUAL_RANGE)
            {
                my_vx -= dx_cursor * CURSOR_AVOID_FACTOR;
                my_vy -= dy_cursor * CURSOR_AVOID_FACTOR;
            }
        }

        // Bias 
        // biased to the right
        if (idx % 419 == 0)
        {
            my_vx = (1 - BIAS) * my_vx + (BIAS * 1);
        }
        // biased to the left
        else if (idx % 409 == 0)
        {
            my_vx = (1 - BIAS) * my_vx + (BIAS * (-1));
        }

        // Edge Avoidance
        if (my_x < EDGE_MARGIN) my_vx += TURN_FACTOR;
        if (my_x > SCREEN_WIDTH - EDGE_MARGIN) my_vx -= TURN_FACTOR;
        if (my_y < EDGE_MARGIN) my_vy += TURN_FACTOR;
        if (my_y > SCREEN_HEIGHT - EDGE_MARGIN) my_vy -= TURN_FACTOR;

        // Speed Limits
        float speed = sqrt(my_vx * my_vx + my_vy * my_vy);
        if (speed < MIN_SPEED)
        {
            my_vx = (my_vx / speed) * MIN_SPEED;
            my_vy = (my_vy / speed) * MIN_SPEED;
        }
        if (speed > MAX_SPEED)
        {
            my_vx = (my_vx / speed) * MAX_SPEED;
            my_vy = (my_vy / speed) * MAX_SPEED;
        }

        my_x += my_vx * dt;
        my_y += my_vy * dt;
        boidsVelocity.vx[idx] = my_vx;
        boidsVelocity.vy[idx] = my_vy;
    }
}

void updateBoidsPositionCPU(float* positions, BoidsVelocity boidsVelocity, int numBoids, float dt)
{
    for (int idx = 0; idx < numBoids; idx++)
    {
        float my_x = ((positions[6 * idx] + 1.0f) / 2.0f) * SCREEN_WIDTH;
        float my_y = (1.0f - ((positions[6 * idx + 1] + 1.0f) / 2.0f)) * SCREEN_HEIGHT;
        my_x += boidsVelocity.vx[idx] * dt;
        my_y += boidsVelocity.vy[idx] * dt;
        positions[6 * idx] = (my_x * 2) / SCREEN_WIDTH - 1.0f;
        positions[6 * idx + 1] = 1.0f - (my_y * 2) / SCREEN_HEIGHT;
    }
}

int oneIterationCPU(float** boids_positions, BoidsVelocity* boidsVelocity)
{
    updateBoidsVelocityCPU(*boids_positions, *boidsVelocity, NUM_BOIDS, DT, CursorOverWindow, cursorX, cursorY);
    updateBoidsPositionCPU(*boids_positions, *boidsVelocity, NUM_BOIDS, DT);
    calculateTriangleVerticesCPU(*boids_positions, *boidsVelocity, NUM_BOIDS);
    return 0;
}



void calculateTriangleVerticesCPU(float* positions, BoidsVelocity boidsVelocity, int num_boids)
{
    for (int boid_index = 0; boid_index < num_boids; boid_index++)
    {
        int index = boid_index * 6;
        if (boid_index >= num_boids) return;

        float my_x = ((positions[index] + 1.0f) / 2.0f) * SCREEN_WIDTH;
        float my_y = (1.0f - ((positions[index + 1] + 1.0f) / 2.0f)) * SCREEN_HEIGHT;

        // Boids are drawn as isosceles triangles of height 5*sqrt(3) heading in direction pointed by point (positions[index], positions[index + 1])
        float s = 2 * sqrtf(3);
        float triangle_h = 5 * sqrtf(3);

        float vector_length = sqrt(boidsVelocity.vx[boid_index] * boidsVelocity.vx[boid_index] + boidsVelocity.vy[boid_index] * boidsVelocity.vy[boid_index]);
        float h_x = my_x - (triangle_h * (boidsVelocity.vx[boid_index] / vector_length));
        float h_y = my_y - (triangle_h * (boidsVelocity.vy[boid_index] / vector_length));

        float x1 = h_x + ((my_y - h_y) / s);
        float x2 = h_x + ((h_y - my_y) / s);
        float y1 = h_y + ((h_x - my_x) / s);
        float y2 = h_y + ((my_x - h_x) / s);

        positions[index] = positions[index];
        positions[index + 2] = (x1 * 2) / SCREEN_WIDTH - 1.0f;
        positions[index + 4] = (x2 * 2) / SCREEN_WIDTH - 1.0f;

        positions[index + 1] = positions[index + 1];
        positions[index + 3] = 1.0f - (y1 * 2) / SCREEN_HEIGHT;
        positions[index + 5] = 1.0f - (y2 * 2) / SCREEN_HEIGHT;
    }
}




