#include "hip/hip_runtime.h"
﻿#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>


#include <stdio.h>
#include <iostream>

#define NUM_BOIDS 5000 
#define BLOCK_SIZE 256
#define VISUAL_RANGE 50.0f
#define PROTECTED_RANGE 10.0f
#define AVOID_FACTOR 0.05f
#define MATCHING_FACTOR 0.05f
#define CENTERING_FACTOR 0.01f
#define MIN_SPEED 2.0f
#define MAX_SPEED 10.0f
#define DT 0.1f
#define TURN_FACTOR 1.0f
#define EDGE_MARGIN 50.0f
#define SCREEN_HEIGHT 600
#define SCREEN_WIDTH 800


struct BoidsVelocity
{
    float* vx, * vy;
};

const char* vertexShaderSource = R"(
#version 330 core
in vec2 position;

void main()
{
    gl_Position = vec4(position, 0.0, 1.0);
})";

const char* fragmentShaderSource = R"(
#version 330 core
out vec4 FragColor;
void main() {
    FragColor = vec4(0.1, 0.6, 0.9, 1.0); // Light blue
}
)";


BoidsVelocity boidsVelocity;
void toNormalised(float* x, float* y, float* norm_x, float* norm_y)
{
    *norm_x = (*x * 2) / SCREEN_WIDTH - 1.0f;
    *norm_y = 1.0f - (*y * 2) / SCREEN_HEIGHT;
}
void fromNormalised(float* x, float* y, float* norm_x, float* norm_y)
{
    *x = ((*norm_x + 1.0f) / 2.0f) * SCREEN_WIDTH;
    *y = (1.0f - ((*norm_y + 1.0f) / 2.0f)) * SCREEN_HEIGHT;
}

__global__ void updateBoids(float* positions, BoidsVelocity boidsVelocity, int numBoids, float dt)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numBoids) return;
    float close_dx = 0, close_dy = 0;
    float xvel_avg = 0, yvel_avg = 0, xpos_avg = 0, ypos_avg = 0;
    int neighbors = 0;

    float my_x = positions[2 * idx];
    float my_y = positions[2 * idx + 1];
    float my_vx = boidsVelocity.vx[idx];
    float my_vy = boidsVelocity.vy[idx];
    //fromNormalised(&my_x, &my_y, &positions[2 * idx], &positions[2 * idx + 1]);
    my_x = ((positions[2 * idx] + 1.0f) / 2.0f) * SCREEN_WIDTH;
    my_y = (1.0f - ((positions[2 * idx + 1] + 1.0f) / 2.0f)) * SCREEN_HEIGHT;

    // Loop through all boids
    for (int i = 0; i < numBoids; i++) {
        if (i == idx) continue;
        float x = ((positions[2 * i] + 1.0f) / 2.0f) * SCREEN_WIDTH;
        float y = (1.0f - ((positions[2 * i + 1] + 1.0f) / 2.0f)) * SCREEN_HEIGHT;
        float dx = x - my_x;
        float dy = y - my_y;
        float dist = sqrt(dx * dx + dy * dy);

        if (dist < PROTECTED_RANGE) { // Separation
            close_dx -= dx;
            close_dy -= dy;
        }
        if (dist < VISUAL_RANGE) { // Alignment and Cohesion
            xvel_avg += boidsVelocity.vx[i];
            yvel_avg += boidsVelocity.vy[i];
            xpos_avg += x;
            ypos_avg += y;
            neighbors++;
        }
    }

    // Calculate alignment and cohesion
    if (neighbors > 0) {
        xvel_avg /= neighbors;
        yvel_avg /= neighbors;
        xpos_avg /= neighbors;
        ypos_avg /= neighbors;

        my_vx += (xvel_avg - my_vx) * MATCHING_FACTOR;   // Alignment
        my_vy += (yvel_avg - my_vy) * MATCHING_FACTOR;

        my_vx += (xpos_avg - my_x) * CENTERING_FACTOR;   // Cohesion
        my_vy += (ypos_avg - my_y) * CENTERING_FACTOR;
    }

    my_vx += close_dx * AVOID_FACTOR;  // Separation
    my_vy += close_dy * AVOID_FACTOR;

    // Edge Avoidance
    if (my_x < EDGE_MARGIN) my_vx += TURN_FACTOR;
    if (my_x > 800 - EDGE_MARGIN) my_vx -= TURN_FACTOR;
    if (my_y < EDGE_MARGIN) my_vy += TURN_FACTOR;
    if (my_y > 600 - EDGE_MARGIN) my_vy -= TURN_FACTOR;

    // Speed Limits
    float speed = sqrt(my_vx * my_vx + my_vy * my_vy);
    if (speed < MIN_SPEED) {
        my_vx = (my_vx / speed) * MIN_SPEED;
        my_vy = (my_vy / speed) * MIN_SPEED;
    }
    if (speed > MAX_SPEED) {
        my_vx = (my_vx / speed) * MAX_SPEED;
        my_vy = (my_vy / speed) * MAX_SPEED;
    }

    // Update position
    my_x += my_vx * dt;
    my_y += my_vy * dt;
    boidsVelocity.vx[idx] = my_vx;
    boidsVelocity.vy[idx] = my_vy;
    //fromNormalised(&my_x, &my_y, &positions[2 * idx], &positions[2 * idx + 1]);
    positions[2 * idx] = (my_x * 2) / SCREEN_WIDTH - 1.0f;
    positions[2 * idx + 1] = 1.0f - (my_y * 2) / SCREEN_HEIGHT;
}

void checkShaderCompilation(GLuint shader, std::string type) {
    GLint success;
    char infoLog[512];
    if (type == "PROGRAM") {
        glGetProgramiv(shader, GL_LINK_STATUS, &success);
        if (!success) {
            glGetProgramInfoLog(shader, 512, NULL, infoLog);
            std::cerr << "ERROR::PROGRAM_LINKING_ERROR: " << infoLog << std::endl;
        }
    }
    else {
        glGetShaderiv(shader, GL_COMPILE_STATUS, &success);
        if (!success) {
            glGetShaderInfoLog(shader, 512, NULL, infoLog);
            std::cerr << "ERROR::SHADER_COMPILATION_ERROR (" << type << "): " << infoLog << std::endl;
        }
    }
}

void initBoids()
{

}

int main()
{
    hipError_t cudaStatus;

    //int deviceCount = 0;
    //hipGetDeviceCount(&deviceCount);
    //std::cout << "CUDA Device Count: " << deviceCount << std::endl;
    //hipDeviceReset();
    //cudaStatus = cudaGLSetGLDevice(0);  // Select the correct GPU
    //if (cudaStatus != hipSuccess) {
    //    std::cerr << "Failed to set CUDA GL Device: " << hipGetErrorString(cudaStatus) << std::endl;
    //    return -1;
    //}


    if (!glfwInit())
    {
        std::cout << "Failed to initialize the GLFW library" << std::endl;
        return -1;
    }
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    GLFWwindow* window = glfwCreateWindow(800, 600, "Shoal of Fish", NULL, NULL);
    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }

    glViewport(0, 0, 800, 600);


    // Shader Compilation
    GLuint vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);
    checkShaderCompilation(vertexShader, "VERTEX");

    GLuint fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);
    checkShaderCompilation(fragmentShader, "FRAGMENT");

    GLuint shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);
    checkShaderCompilation(shaderProgram, "PROGRAM");

    glDeleteShader(vertexShader);
    glDeleteShader(fragmentShader);


    GLuint VBO, VAO;
    hipGraphicsResource* cudaVBO;
    float* temp_positions = (float*)malloc(NUM_BOIDS * 2 * sizeof(float));

    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glBindVertexArray(VAO);


    for (int i = 0; i < NUM_BOIDS * 2; i += 2) {
        //std::cout << cudaVBO << " ";

        temp_positions[i] = ((rand() % 800) / 400.0f) - 1.0f; // Normalize X to [-1, 1]
        temp_positions[i + 1] = ((rand() % 600) / 300.0f) - 1.0f; // Normalize Y to [-1, 1]
        /*((float*)cudaVBO)[i] = rand() % 800;
        ((float*)cudaVBO)[i + 1] = rand() % 600;*/
        //std::cout << &cudaVBO << " ";
    }
    
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, NUM_BOIDS * 2 * sizeof(float), temp_positions, GL_DYNAMIC_DRAW);

    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 0, 0);
    glEnableVertexAttribArray(0);

    cudaStatus = hipGraphicsGLRegisterBuffer(&cudaVBO, VBO, hipGraphicsRegisterFlagsNone);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Error registering buffer with CUDA: " << hipGetErrorString(cudaStatus) << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc(&boidsVelocity.vx, NUM_BOIDS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return -1;
    }
    cudaStatus = hipMalloc(&boidsVelocity.vy, NUM_BOIDS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return -1;
    }

    float* temp_vx = (float*)malloc(NUM_BOIDS * sizeof(float));
    float* temp_vy = (float*)malloc(NUM_BOIDS * sizeof(float));

    //cudaStatus = hipGraphicsMapResources(1, &cudaVBO, 0);
    //if (cudaStatus != hipSuccess) {
    //    std::cerr << "Error mapping CUDA resource!" << std::endl;
    //    return -1;  
    //}
    //size_t size;
    //cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&temp_positions, &size, cudaVBO);
    //if (cudaStatus != hipSuccess) {
    //    std::cerr << "Error getting mapped pointer!" << std::endl;
    //    return -1;  
    //}
    //for (int i = 0; i < NUM_BOIDS * 2; i += 2) {
    //    //std::cout << cudaVBO << " ";
    //    
    //    ((float*)temp_positions)[i] = rand() % 800;
    //    ((float*)temp_positions)[i + 1] = rand() % 600;
    //    /*((float*)cudaVBO)[i] = rand() % 800;
    //    ((float*)cudaVBO)[i + 1] = rand() % 600;*/
    //    //std::cout << &cudaVBO << " ";
    //}
    //cudaStatus = hipGraphicsUnmapResources(1, (hipGraphicsResource**)&temp_positions, 0);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipGraphicsUnmapResources failed!");
    //    return -1;
    //}
    for (int i = 0; i < NUM_BOIDS; i++) {
        temp_vx[i] = ((rand() % 20) - 10) / 10.0f;
        temp_vy[i] = ((rand() % 20) - 10) / 10.0f;
    }
    

    cudaStatus = hipMemcpy(boidsVelocity.vx, temp_vx, NUM_BOIDS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return -1;
    }
    cudaStatus = hipMemcpy(boidsVelocity.vy, temp_vy, NUM_BOIDS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return -1;
    }
    free(temp_vx);
    free(temp_vy);
    free(temp_positions);

    glPointSize(3.0f);
    while (!glfwWindowShouldClose(window))
    {

        hipGraphicsMapResources(1, &cudaVBO, 0);
        cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&temp_positions, NULL, cudaVBO);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsResourceGetMappedPointer launch failed: %s\n", hipGetErrorString(cudaStatus));
            return -1;
        }

        // Update boids
        updateBoids << <(NUM_BOIDS + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >> > (temp_positions, boidsVelocity, NUM_BOIDS, DT);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
        }
        hipDeviceSynchronize();

        cudaStatus = hipGraphicsUnmapResources(1, &cudaVBO, 0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGraphicsResourceGetMappedPointer launch failed: %s\n", hipGetErrorString(cudaStatus));
            return -1;
        }

        // Render
        glClear(GL_COLOR_BUFFER_BIT);

        glUseProgram(shaderProgram);
        glBindVertexArray(VAO);
        glDrawArrays(GL_POINTS, 0, NUM_BOIDS);

        glfwSwapBuffers(window);
        glfwPollEvents();

    }



    hipGraphicsUnregisterResource(cudaVBO);
    glDeleteBuffers(1, &VBO);
    glDeleteVertexArrays(1, &VAO);
    hipFree(boidsVelocity.vx);
    hipFree(boidsVelocity.vy);
    glDeleteProgram(shaderProgram);
    glfwTerminate();
    

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}




