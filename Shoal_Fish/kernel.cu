#include "hip/hip_runtime.h"
﻿#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>


#include <stdio.h>
#include <iostream>

#define NUM_BOIDS 10000
#define BLOCK_SIZE 256
#define VISUAL_RANGE 50.0f
#define PROTECTED_RANGE 10.0f
#define AVOID_FACTOR 0.05f
#define MATCHING_FACTOR 0.05f
#define CENTERING_FACTOR 0.01f
#define MIN_SPEED 2.0f
#define MAX_SPEED 10.0f
#define DT 0.1f
#define TURN_FACTOR 1.0f
#define EDGE_MARGIN 100.0f


struct BoidsVelocity
{
    float* vx, * vy;
};

const char* vertexShaderSource = R"(
#version 330 core
in vec2 position;

void main()
{
    gl_Position = vec4(position, 0.0, 1.0);
})";

const char* fragmentShaderSource = R"(
#version 330 core
out vec4 FragColor;
void main() {
    FragColor = vec4(0.1, 0.6, 0.9, 1.0); // Light blue
}
)";


BoidsVelocity boidsVelocity;

__global__ void updateBoids(float* positions, BoidsVelocity boidsVelocity, int numBoids, float dt)
{
    return;
}

void checkShaderCompilation(GLuint shader, std::string type) {
    GLint success;
    char infoLog[512];
    if (type == "PROGRAM") {
        glGetProgramiv(shader, GL_LINK_STATUS, &success);
        if (!success) {
            glGetProgramInfoLog(shader, 512, NULL, infoLog);
            std::cerr << "ERROR::PROGRAM_LINKING_ERROR: " << infoLog << std::endl;
        }
    }
    else {
        glGetShaderiv(shader, GL_COMPILE_STATUS, &success);
        if (!success) {
            glGetShaderInfoLog(shader, 512, NULL, infoLog);
            std::cerr << "ERROR::SHADER_COMPILATION_ERROR (" << type << "): " << infoLog << std::endl;
        }
    }
}

void initBoids()
{

}

int main()
{
    hipError_t cudaStatus;

    //int deviceCount = 0;
    //hipGetDeviceCount(&deviceCount);
    //std::cout << "CUDA Device Count: " << deviceCount << std::endl;
    //hipDeviceReset();
    //cudaStatus = cudaGLSetGLDevice(0);  // Select the correct GPU
    //if (cudaStatus != hipSuccess) {
    //    std::cerr << "Failed to set CUDA GL Device: " << hipGetErrorString(cudaStatus) << std::endl;
    //    return -1;
    //}


    if (!glfwInit())
    {
        std::cout << "Failed to initialize the GLFW library" << std::endl;
        return -1;
    }
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    GLFWwindow* window = glfwCreateWindow(800, 600, "Shoal of Fish", NULL, NULL);
    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }

    glViewport(0, 0, 800, 600);


    // Shader Compilation
    GLuint vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);
    checkShaderCompilation(vertexShader, "VERTEX");

    GLuint fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);
    checkShaderCompilation(fragmentShader, "FRAGMENT");

    GLuint shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);
    checkShaderCompilation(shaderProgram, "PROGRAM");

    glDeleteShader(vertexShader);
    glDeleteShader(fragmentShader);


    GLuint VBO, VAO;
    hipGraphicsResource* cudaVBO;
    float* temp_positions = (float*)malloc(NUM_BOIDS * 2 * sizeof(float));

    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glBindVertexArray(VAO);


    for (int i = 0; i < NUM_BOIDS * 2; i += 2) {
        //std::cout << cudaVBO << " ";

        temp_positions[i] = ((rand() % 800) / 400.0f) - 1.0f; // Normalize X to [-1, 1]
        temp_positions[i + 1] = ((rand() % 600) / 300.0f) - 1.0f; // Normalize Y to [-1, 1]
        /*((float*)cudaVBO)[i] = rand() % 800;
        ((float*)cudaVBO)[i + 1] = rand() % 600;*/
        //std::cout << &cudaVBO << " ";
    }
    
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, NUM_BOIDS * 2 * sizeof(float), temp_positions, GL_DYNAMIC_DRAW);

    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 0, 0);
    glEnableVertexAttribArray(0);

    cudaStatus = hipGraphicsGLRegisterBuffer(&cudaVBO, VBO, hipGraphicsRegisterFlagsNone);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Error registering buffer with CUDA: " << hipGetErrorString(cudaStatus) << std::endl;
        return -1;
    }

    cudaStatus = hipMalloc(&boidsVelocity.vx, NUM_BOIDS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return -1;
    }
    cudaStatus = hipMalloc(&boidsVelocity.vy, NUM_BOIDS * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return -1;
    }

    float* temp_vx = (float*)malloc(NUM_BOIDS * sizeof(float));
    float* temp_vy = (float*)malloc(NUM_BOIDS * sizeof(float));

    //cudaStatus = hipGraphicsMapResources(1, &cudaVBO, 0);
    //if (cudaStatus != hipSuccess) {
    //    std::cerr << "Error mapping CUDA resource!" << std::endl;
    //    return -1;  
    //}
    //size_t size;
    //cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&temp_positions, &size, cudaVBO);
    //if (cudaStatus != hipSuccess) {
    //    std::cerr << "Error getting mapped pointer!" << std::endl;
    //    return -1;  
    //}
    //for (int i = 0; i < NUM_BOIDS * 2; i += 2) {
    //    //std::cout << cudaVBO << " ";
    //    
    //    ((float*)temp_positions)[i] = rand() % 800;
    //    ((float*)temp_positions)[i + 1] = rand() % 600;
    //    /*((float*)cudaVBO)[i] = rand() % 800;
    //    ((float*)cudaVBO)[i + 1] = rand() % 600;*/
    //    //std::cout << &cudaVBO << " ";
    //}
    //cudaStatus = hipGraphicsUnmapResources(1, (hipGraphicsResource**)&temp_positions, 0);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipGraphicsUnmapResources failed!");
    //    return -1;
    //}
    for (int i = 0; i < NUM_BOIDS; i++) {
        temp_vx[i] = ((rand() % 20) - 10) / 10.0f;
        temp_vy[i] = ((rand() % 20) - 10) / 10.0f;
    }
    

    cudaStatus = hipMemcpy(boidsVelocity.vx, temp_vx, NUM_BOIDS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return -1;
    }
    cudaStatus = hipMemcpy(boidsVelocity.vy, temp_vy, NUM_BOIDS * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return -1;
    }
    free(temp_vx);
    free(temp_vy);
    free(temp_positions);

    while (!glfwWindowShouldClose(window))
    {

        hipGraphicsMapResources(1, &cudaVBO, 0);
        hipGraphicsResourceGetMappedPointer((void**)&temp_positions, NULL, cudaVBO);

        // Update boids
        updateBoids << <(NUM_BOIDS + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >> > (temp_positions, boidsVelocity, NUM_BOIDS, DT);

        hipGraphicsUnmapResources(1, &cudaVBO, 0);

        // Render
        glClear(GL_COLOR_BUFFER_BIT);

        glUseProgram(shaderProgram);
        glBindVertexArray(VAO);
        glDrawArrays(GL_POINTS, 0, NUM_BOIDS);

        glfwSwapBuffers(window);
        glfwPollEvents();

    }



    hipGraphicsUnregisterResource(cudaVBO);
    glDeleteBuffers(1, &VBO);
    glDeleteVertexArrays(1, &VAO);
    hipFree(boidsVelocity.vx);
    hipFree(boidsVelocity.vy);
    glDeleteProgram(shaderProgram);
    glfwTerminate();
    

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


    /*cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }*/

    // Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    /*cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }*/
   



